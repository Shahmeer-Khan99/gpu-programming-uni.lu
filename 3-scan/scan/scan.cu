#include "hip/hip_runtime.h"
#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  n++;
  return n;
}

void printDeviceArray(int* device_array, int N, const char* label) {
    int* host_array = new int[N];
    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << label << ": ";
    for (int i = 0; i < N; i++) {
        std::cout << host_array[i] << " ";
    }
    std::cout << std::endl;
    delete[] host_array;
}

__global__ void downsweep(int* result,  int N, int stride) {    
  int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);
  int jump = stride * 2;
  int leftIndex = threadIndex * jump + stride - 1;
  int rightIndex = threadIndex * jump + jump - 1;
  int splits = N / jump;
  if(threadIndex < splits) {
    int temp = result[rightIndex];
    result[rightIndex] += result[leftIndex];
    result[leftIndex] = temp;
  }
}

__global__ void upsweep(int* result,  int N, int stride) {
  int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);
  int jump = stride * 2;
  int leftIndex = threadIndex * jump + stride - 1;
  int rightIndex = threadIndex * jump + jump - 1;
  int splits = N / jump;
  if(threadIndex < splits) {
    result[rightIndex] += result[leftIndex];
  }
}


void exclusive_scan(int* input, int N, int* result) {

  for(int stride = 1; stride < N; stride *= 2) {
    int strided = 2 * stride;
    int num_blocks = (N / strided + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // int num_blocks = N / strided;
    upsweep<<<num_blocks, THREADS_PER_BLOCK>>>(result, N, stride);
    hipDeviceSynchronize();
  }

  int zero = 0;
  hipMemcpy(result + N - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

  for(int stride = N/2; stride > 0; stride /= 2) {
    int strided = 2 * stride;
    int num_blocks = (N / strided + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // int num_blocks = N / strided;
    downsweep<<<num_blocks, THREADS_PER_BLOCK>>>(result, N, stride);
    hipDeviceSynchronize();
  }
}

double cudaScan(int* inarray, int* end, int* resultarray)
{
  int* device_result;
  int* device_input;
  int N = end - inarray;  

  int rounded_length = nextPow2(end - inarray);
  
  hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
  hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

  hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

  double startTime = CycleTimer::currentSeconds();
  exclusive_scan(device_input, rounded_length, device_result);

  hipDeviceSynchronize();
  double endTime = CycleTimer::currentSeconds();
      
  hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

  double overallDuration = endTime - startTime;
  return overallDuration; 
}

double cudaScanThrust(int* inarray, int* end, int* resultarray) {
  int length = end - inarray;
  thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
  thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
  
  hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

  double startTime = CycleTimer::currentSeconds();

  thrust::exclusive_scan(d_input, d_input + length, d_output);

  hipDeviceSynchronize();
  double endTime = CycleTimer::currentSeconds();
  
  hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

  thrust::device_free(d_input);
  thrust::device_free(d_output);

  double overallDuration = endTime - startTime;
  return overallDuration; 
}


__global__ void map_repeats(int* input, int N, int* output) {
  int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);
  if(threadIndex < N - 1) {
    if(input[threadIndex] == input[threadIndex + 1]) {
      output[threadIndex] = 1;
    } else {
      output[threadIndex] = 0;
    }
  }
  if(threadIndex == N - 1) {
      output[threadIndex] = 0;
  }
}


__global__ void get_repeats(int* scan, int* output, int* flags, int length) {
  int threadIndex = threadIdx.x + (blockDim.x * blockIdx.x);
  if(threadIndex < length - 1 && flags[threadIndex] == 1) {
    output[scan[threadIndex]] = threadIndex;
  }
}


int find_repeats(int* device_input, int length, int* device_output) {
  int* device_flags;
  int* device_scan;
  int rounded_length = nextPow2(length);

  hipMalloc((void **)&device_flags, sizeof(int) * rounded_length);
  hipMalloc((void **)&device_scan, sizeof(int) * rounded_length);

  printDeviceArray(device_input, length, "DEVICE INPUT");

  // hipMemcpy(device_input, device, N * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(device_result, inarray, N * sizeof(int), hipMemcpyHostToDevice);
  int blocks = (length + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
  map_repeats<<<blocks, THREADS_PER_BLOCK>>>(device_input, length, device_flags);
  hipDeviceSynchronize();

  printDeviceArray(device_flags, length, "DEVICE MAP");

  hipMemcpy(device_scan, device_flags, length * sizeof(int), hipMemcpyHostToDevice);

  exclusive_scan(device_flags, rounded_length, device_scan);
  hipDeviceSynchronize();

  printDeviceArray(device_scan, length, "DEVICE SCANNN");

  int total_repeats;
  hipMemcpy(&total_repeats,  device_scan + length - 1, sizeof(int), hipMemcpyDeviceToHost);

  get_repeats<<<blocks, THREADS_PER_BLOCK>>>(device_scan, device_output, device_flags, length);
  hipDeviceSynchronize();

  hipFree(device_flags);
  hipFree(device_scan);
  
  return total_repeats;
}

double cudaFindRepeats(int *input, int length, int *output, int *output_length) {
  int *device_input;
  int *device_output;
  int rounded_length = nextPow2(length);
  
  hipMalloc((void **)&device_input, rounded_length * sizeof(int));
  hipMalloc((void **)&device_output, rounded_length * sizeof(int));
  hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  double startTime = CycleTimer::currentSeconds();
  
  int result = find_repeats(device_input, length, device_output);

  hipDeviceSynchronize();
  double endTime = CycleTimer::currentSeconds();

  // set output count and results array
  *output_length = result;
  hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(device_input);
  hipFree(device_output);

  float duration = endTime - startTime; 
  return duration;
}



void printCudaInfo()
{
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i=0; i<deviceCount; i++)
  {
      hipDeviceProp_t deviceProps;
      hipGetDeviceProperties(&deviceProps, i);
      printf("Device %d: %s\n", i, deviceProps.name);
      printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
      printf("   Global mem: %.0f MB\n",
              static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
      printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n"); 
}
